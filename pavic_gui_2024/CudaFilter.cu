#include "hip/hip_runtime.h"
#include "CudaFilter.h"
#include ""
#include <stdio.h>

//====================================================================
// FILTRO S�PIA (c�digo existente, sem altera��es)
//====================================================================
__device__ unsigned char clamp(float value) {
    if (value > 255.0f) return 255;
    return (unsigned char)value;
}

__global__ void sepia_kernel(unsigned char* out, const unsigned char* in, int width, int height, int stride)
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (col < width && row < height)
    {
        int pixel_offset = row * stride + col * 3;
        if (pixel_offset + 2 < stride * height) {
            float b = in[pixel_offset];
            float g = in[pixel_offset + 1];
            float r = in[pixel_offset + 2];
            float new_r = r * 0.393f + g * 0.769f + b * 0.189f;
            float new_g = r * 0.349f + g * 0.686f + b * 0.168f;
            float new_b = r * 0.272f + g * 0.534f + b * 0.131f;
            out[pixel_offset] = clamp(new_b);
            out[pixel_offset + 1] = clamp(new_g);
            out[pixel_offset + 2] = clamp(new_r);
        }
    }
}

extern "C" void run_sepia_filter_cuda(unsigned char* h_output_image, const unsigned char* h_input_image, int width, int height, int stride)
{
    unsigned char* d_in, * d_out;
    int image_size_bytes = stride * height;
    CUDA_CHECK(hipMalloc((void**)&d_in, image_size_bytes));
    CUDA_CHECK(hipMalloc((void**)&d_out, image_size_bytes));
    CUDA_CHECK(hipMemcpy(d_in, h_input_image, image_size_bytes, hipMemcpyHostToDevice));
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((width + threadsPerBlock.x - 1) / threadsPerBlock.x, (height + threadsPerBlock.y - 1) / threadsPerBlock.y);
    sepia_kernel<<<numBlocks, threadsPerBlock>>>(d_out, d_in, width, height, stride);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
    CUDA_CHECK(hipMemcpy(h_output_image, d_out, image_size_bytes, hipMemcpyDeviceToHost));
    CUDA_CHECK(hipFree(d_in));
    CUDA_CHECK(hipFree(d_out));
}

//====================================================================
// NOVO: FILTRO DE INVERS�O DE CORES
//====================================================================

// Kernel CUDA que inverte a cor de cada pixel
__global__ void inversion_kernel(unsigned char* out, const unsigned char* in, int width, int height, int stride)
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (col < width && row < height)
    {
        int pixel_offset = row * stride + col * 3;
        if (pixel_offset + 2 < stride * height) {
            // A l�gica � simples: 255 - valor_original para cada canal
            out[pixel_offset] = 255 - in[pixel_offset];     // Blue
            out[pixel_offset + 1] = 255 - in[pixel_offset + 1]; // Green
            out[pixel_offset + 2] = 255 - in[pixel_offset + 2]; // Red
        }
    }
}

// Fun��o Wrapper para o filtro de invers�o
extern "C" void run_inversion_filter_cuda(unsigned char* h_output_image, const unsigned char* h_input_image, int width, int height, int stride)
{
    unsigned char* d_in, * d_out;
    int image_size_bytes = stride * height;
    CUDA_CHECK(hipMalloc((void**)&d_in, image_size_bytes));
    CUDA_CHECK(hipMalloc((void**)&d_out, image_size_bytes));
    CUDA_CHECK(hipMemcpy(d_in, h_input_image, image_size_bytes, hipMemcpyHostToDevice));
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((width + threadsPerBlock.x - 1) / threadsPerBlock.x, (height + threadsPerBlock.y - 1) / threadsPerBlock.y);

    // Lan�a o novo kernel de invers�o
    inversion_kernel<<<numBlocks, threadsPerBlock>>>(d_out, d_in, width, height, stride);

    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
    CUDA_CHECK(hipMemcpy(h_output_image, d_out, image_size_bytes, hipMemcpyDeviceToHost));
    CUDA_CHECK(hipFree(d_in));
    CUDA_CHECK(hipFree(d_out));
}
